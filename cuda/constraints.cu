#include <cuda/constraints.hh>
#include <cuda/kernels.hh>
#include <cuda/helper.hh>
#include <string.h>
#include <stdio.h>
#include <assert.h>

#define CSTR_VAL_2X     CSTR_NO,  CSTR_NO
#define CSTR_VAL_4X     CSTR_VAL_2X,  CSTR_VAL_2X
#define CSTR_VAL_8X     CSTR_VAL_4X,  CSTR_VAL_4X
#define CSTR_VAL_16X    CSTR_VAL_8X,  CSTR_VAL_8X
#define CSTR_VAL_32X    CSTR_VAL_16X, CSTR_VAL_16X
#define CSTR_VAL_64X    CSTR_VAL_32X, CSTR_VAL_32X
#define CSTR_VAL_128X   CSTR_VAL_64X, CSTR_VAL_64X
#define CSTR_VAL_256X   CSTR_VAL_128X, CSTR_VAL_128X
#define CSTR_VAL_512X   CSTR_VAL_256X, CSTR_VAL_256X


CUDA_DEVICE __constant__ uintptr_t  cstrData[CSTR_MAX_CSTR * 8] = {CSTR_VAL_512X, CSTR_VAL_512X};
CUDA_DEVICE __constant__ TVarType   cstrType[CSTR_MAX_VAR];
CUDA_DEVICE __constant__ Gecode::TQuantifier cstrQuan[CSTR_MAX_VAR];
CUDA_DEVICE __constant__ int        cstrDom[CSTR_MAX_VAR];
CUDA_DEVICE __constant__ size_t     cstrPoly[CSTR_MAX_POLY];
                         size_t     cstrPolyNext = 0;
CUDA_DEVICE __constant__ size_t     cstrVarNumberD = 0;
                         size_t     cstrVarNumberH = 0;
                         size_t     cstrDomSize = 0;

CUDA_DEVICE cstrFuncPtr     cstrTable[64] = {
        &cstrEq,       NULL,          NULL,          NULL,
        NULL,          NULL,          NULL,          NULL,

        &cstrAndNQ,    &cstrAndEQ,    &cstrAndLQ,    &cstrAndLE,
        &cstrAndGQ,    &cstrAndGR,    NULL,          NULL,

        &cstrOrNQ,     &cstrOrEQ,     &cstrOrLQ,     &cstrOrLE,
        &cstrOrGQ,     &cstrOrGR,     NULL,          NULL,

        &cstrImpNQ,    &cstrImpEQ,    &cstrImpLQ,    &cstrImpLE,
        &cstrImpGQ,    &cstrImpGR,    NULL,          NULL,

        &cstrXorNQ,    &cstrXorEQ,    &cstrXorLQ,    &cstrXorLE,
        &cstrXorGQ,    &cstrXorGR,    NULL,          NULL,

        &cstrPlusNQ,   &cstrPlusEQ,   &cstrPlusLQ,   &cstrPlusLE,
        &cstrPlusGQ,   &cstrPlusGR,   NULL,          NULL,

        &cstrTimesNQ,  &cstrTimesEQ,  &cstrTimesLQ,  &cstrTimesLE,
        &cstrTimesGQ,  &cstrTimesGR,  NULL,          NULL,

        &cstrLinearNQ, &cstrLinearEQ, &cstrLinearLQ, &cstrLinearLE,
        &cstrLinearGQ, &cstrLinearGR, NULL,          NULL
};

CUDA_HOST   size_t pushPolyToGPU(size_t * poly, size_t size) {
    size_t next = cstrPolyNext;
    assert(next + 2 * size < CSTR_MAX_POLY);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrPoly), poly, size * sizeof(size_t), next * sizeof(size_t)));
    cstrPolyNext += 2 * size;

    return next;
}

CUDA_HOST   void pushVarToGPU(TVarType * type, Gecode::TQuantifier * quant, size_t size) {
    assert(size < CSTR_MAX_VAR);
    assert(type != nullptr);
    assert(quant != nullptr);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrVarNumberD), &cstrVarNumberH, size, sizeof(size_t)));
    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrType), type, size * sizeof(TVarType)));
    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrQuan), quant, size * sizeof(Gecode::TQuantifier)));

    cstrVarNumberH = size;
}

CUDA_HOST void pushDomToGPU(int * dom, size_t size) {
    assert(size < CSTR_MAX_VAR);
    assert(size == 2 * cstrVarNumberH);
    assert(dom != nullptr);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrDom), dom, size * sizeof(int)));

    cstrDomSize = 0;

    for (size_t i = 0; i < size; i += 2) {
        cstrDomSize += (dom[i + 1] - dom[i]);
    }
}

CUDA_HOST void pushCstrToGPU(uintptr_t * cstrs, size_t size) {
    assert(size < (CSTR_MAX_CSTR * 8));
    assert(cstrs != nullptr);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrData), cstrs, size * sizeof(uintptr_t)));
}

CUDA_HOST   int *   initPopulation(size_t popSize, size_t indSize) {
    dim3 grid, block;
    int * d_pop;

    CCR(hipMalloc((void**)&d_pop, sizeof(int) * cstrVarNumberH * popSize * indSize));
    initPopulationKernel<<<grid, block>>>(d_pop, popSize, indSize);
    CCR(hipGetLastError());

    return d_pop;
}

CUDA_HOST   void    doTheMagic(int * pop, size_t popSize, size_t indSize, size_t gen) {
    dim3 grid, block;

    assert(pop != nullptr);

    doTheMagicKernel<<<grid, block>>>(pop, popSize, indSize, gen);
    CCR(hipGetLastError());
}

CUDA_HOST   size_t*    getResults(int * pop, size_t popSize, size_t indSize) {
    dim3 grid, block;
    static size_t * d_res=  nullptr;
    size_t * h_res = nullptr;
    static size_t domSize = 0;

    assert(pop != nullptr);

    if (domSize == 0 || cstrDomSize != domSize) {
        domSize = cstrDomSize;

        if (d_res) {
            CCR(hipFree((void*)d_res));
        }

        CCR(hipMalloc((void**)&d_res, sizeof(size_t) * domSize));
    }

    getResultsKernel<<<grid, block>>>(pop, popSize, indSize, cstrDomSize, d_res);
    CCR(hipGetLastError());
    CCR(hipFree((void*)pop));

    h_res = new size_t[cstrDomSize];

    CCR(hipMemcpy(h_res, d_res, sizeof(size_t) * cstrDomSize, hipMemcpyDeviceToHost));

    return h_res;
}

CUDA_GLOBAL void    initPopulationKernel(int * popPtr, size_t popSize, size_t indSize) {}
CUDA_GLOBAL void    doTheMagicKernel(int * pop, size_t popSize, size_t indSize, size_t gen) {}

CUDA_GLOBAL void    getResultsKernel(int * pop, size_t popSize, size_t indSize, size_t domSize, size_t* res) {
    size_t  gtid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t  sum = 0;
    size_t  idx = 0;
    int     val = cstrDom[0];

    if (gtid < domSize) {
        for (size_t i = 0; (i < domSize) && (i < gtid); ++i) {
            ++val;

            if (val > cstrDom[2 * idx + 1]) {
                ++idx;
                val = cstrDom[2 * idx];
            }
        }

        for (size_t i = 0; i < popSize; ++i) {
            sum += (pop[i * indSize + idx] == val);
        }
    }
}

CUDA_DEVICE bool cstrValidate(int * c) {
    for (size_t i = 0; cstrData[8 * i] != CSTR_NO && (8 * i) < CSTR_MAX_CSTR; ++i) {
        if (!cstrTable[cstrData[8 * i]](cstrData + (8 * i) + 1, c)) {
            return false;
        }
    }
    return true;
}

CUDA_DEVICE bool cstrEq(uintptr_t * data, int * c) {
    size_t v0 = (size_t) data[0];
    int    val = uint2int((unsigned int) data[1]);

    return c[v0] == val;
}

CUDA_DEVICE bool cstrAndEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opAnd(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrAndNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opAnd(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrAndGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opAnd(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrAndGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opAnd(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrAndLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opAnd(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrAndLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opAnd(p0, c[v0], p1, c[v1]) <= c[v2];
}


CUDA_DEVICE bool cstrOrEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opOr(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrOrNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opOr(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrOrGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opOr(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrOrGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opOr(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrOrLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opOr(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrOrLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opOr(p0, c[v0], p1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrImpEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opImp(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrImpNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opImp(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrImpGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opImp(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrImpGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opImp(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrImpLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opImp(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrImpLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opImp(p0, c[v0], p1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrXorEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opXor(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrXorNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opXor(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrXorGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opXor(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrXorGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opXor(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrXorLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opXor(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrXorLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opXor(p0, c[v0], p1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrPlusEQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opPlus(n0, c[v0], n1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrPlusNQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opPlus(n0, c[v0], n1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrPlusGQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opPlus(n0, c[v0], n1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrPlusGR(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opPlus(n0, c[v0], n1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrPlusLQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opPlus(n0, c[v0], n1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrPlusLE(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return opPlus(n0, c[v0], n1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrTimesEQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return opTimes(n, c[v0], c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrTimesNQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return opTimes(n, c[v0], c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrTimesGQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return opTimes(n, c[v0], c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrTimesGR(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return opTimes(n, c[v0], c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrTimesLQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return opTimes(n, c[v0], c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrTimesLE(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return opTimes(n, c[v0], c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrLinearEQ(uintptr_t * data, int * c) {
    size_t vIdx = (size_t) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    size_t *v = cstrPoly + vIdx;
    int sum = 0;

    opLinear(v, size, sum);
    return sum == c[v0];
}

CUDA_DEVICE bool cstrLinearNQ(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    opLinear(v, size, sum);
    return sum != c[v0];
}

CUDA_DEVICE bool cstrLinearGQ(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    opLinear(v, size, sum);
    return sum > c[v0];
}

CUDA_DEVICE bool cstrLinearGR(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    opLinear(v, size, sum);
    return sum >= c[v0];
}

CUDA_DEVICE bool cstrLinearLQ(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    opLinear(v, size, sum);
    return sum < c[v0];
}

CUDA_DEVICE bool cstrLinearLE(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    opLinear(v, size, sum);
    return sum <= c[v0];
}
