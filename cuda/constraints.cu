#include <cuda/constraints.hh>
#include <cuda/kernels.hh>
#include <cuda/helper.hh>
#include <string.h>
#include <stdio.h>
#include <assert.h>
#include <hiprand/hiprand_kernel.h>

#define CSTR_VAL_2X     CSTR_NO,  CSTR_NO
#define CSTR_VAL_4X     CSTR_VAL_2X,  CSTR_VAL_2X
#define CSTR_VAL_8X     CSTR_VAL_4X,  CSTR_VAL_4X
#define CSTR_VAL_16X    CSTR_VAL_8X,  CSTR_VAL_8X
#define CSTR_VAL_32X    CSTR_VAL_16X, CSTR_VAL_16X
#define CSTR_VAL_64X    CSTR_VAL_32X, CSTR_VAL_32X
#define CSTR_VAL_128X   CSTR_VAL_64X, CSTR_VAL_64X
#define CSTR_VAL_256X   CSTR_VAL_128X, CSTR_VAL_128X
#define CSTR_VAL_512X   CSTR_VAL_256X, CSTR_VAL_256X


CUDA_DEVICE __constant__ uintptr_t  cstrData[CSTR_MAX_CSTR * 8] = {CSTR_VAL_512X, CSTR_VAL_512X};
CUDA_DEVICE __constant__ TVarType   cstrType[CSTR_MAX_VAR];
CUDA_DEVICE __constant__ Gecode::TQuantifier cstrQuan[CSTR_MAX_VAR];
CUDA_DEVICE __constant__ int        cstrDom[CSTR_MAX_VAR];
CUDA_DEVICE __constant__ size_t     cstrPoly[CSTR_MAX_POLY];
                         size_t     cstrPolyNext = 0;
                         size_t     cstrVarNumber = 0;
                         size_t     cstrDomSize = 0;

CUDA_DEVICE              hiprandState_t *cstrRandStates = nullptr;

CUDA_DEVICE cstrFuncPtr     cstrTable[64] = {
        &cstrEq,       NULL,          NULL,          NULL,
        NULL,          NULL,          NULL,          NULL,

        &cstrAndNQ,    &cstrAndEQ,    &cstrAndLQ,    &cstrAndLE,
        &cstrAndGQ,    &cstrAndGR,    NULL,          NULL,

        &cstrOrNQ,     &cstrOrEQ,     &cstrOrLQ,     &cstrOrLE,
        &cstrOrGQ,     &cstrOrGR,     NULL,          NULL,

        &cstrImpNQ,    &cstrImpEQ,    &cstrImpLQ,    &cstrImpLE,
        &cstrImpGQ,    &cstrImpGR,    NULL,          NULL,

        &cstrXorNQ,    &cstrXorEQ,    &cstrXorLQ,    &cstrXorLE,
        &cstrXorGQ,    &cstrXorGR,    NULL,          NULL,

        &cstrPlusNQ,   &cstrPlusEQ,   &cstrPlusLQ,   &cstrPlusLE,
        &cstrPlusGQ,   &cstrPlusGR,   NULL,          NULL,

        &cstrTimesNQ,  &cstrTimesEQ,  &cstrTimesLQ,  &cstrTimesLE,
        &cstrTimesGQ,  &cstrTimesGR,  NULL,          NULL,

        &cstrLinearNQ, &cstrLinearEQ, &cstrLinearLQ, &cstrLinearLE,
        &cstrLinearGQ, &cstrLinearGR, NULL,          NULL
};

CUDA_HOST   size_t pushPolyToGPU(size_t * poly, size_t size) {
    size_t next = cstrPolyNext;
    assert(next + 2 * size < CSTR_MAX_POLY);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrPoly), poly, size * sizeof(size_t), next * sizeof(size_t)));
    cstrPolyNext += 2 * size;

    return next;
}

CUDA_HOST   void pushVarToGPU(TVarType * type, Gecode::TQuantifier * quant, size_t size) {
    assert(size < CSTR_MAX_VAR);
    assert(type != nullptr);
    assert(quant != nullptr);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrType), type, size * sizeof(TVarType)));
    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrQuan), quant, size * sizeof(Gecode::TQuantifier)));

    cstrVarNumber = size;
}

CUDA_HOST void pushDomToGPU(int * dom, size_t size) {
    assert(size < CSTR_MAX_VAR);
    assert(size == 2 * cstrVarNumber);
    assert(dom != nullptr);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrDom), dom, size * sizeof(int)));

    cstrDomSize = 0;

    for (size_t i = 0; i < size; i += 2) {
        cstrDomSize += (dom[i + 1] - dom[i]);
    }
}

CUDA_HOST void pushCstrToGPU(uintptr_t * cstrs, size_t size) {
    assert(size < (CSTR_MAX_CSTR * 8));
    assert(cstrs != nullptr);

    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrData), cstrs, size * sizeof(uintptr_t)));
}

/**
 * Calls the kernel which initializes the population
 * @param popSize number of individuals in the population
 * @param indSize number of variables in an individual
 * @return initialized population
 */
CUDA_HOST   int *   initPopulation(size_t popSize, size_t indSize) {
    dim3 grid, block;
    int * d_pop;
    hiprandState_t *state;

    block = dim3(BLOCK_SIZE);
    grid = dim3((popSize + BLOCK_SIZE - 1)/ BLOCK_SIZE);

    CCR(hipMalloc((void**)&d_pop, sizeof(int) * popSize * indSize));
    CCR(hipMalloc((void**)&state, sizeof(hiprandState_t) * popSize));
    CCR(hipMemcpyToSymbol(HIP_SYMBOL(cstrRandStates), &state, sizeof(hiprandState_t*)));
    initPopulationKernel<<<grid, block>>>(d_pop, popSize, indSize);
    CCR(hipGetLastError());

    return d_pop;
}

CUDA_HOST   void    doTheMagic(int * pop, size_t popSize, size_t indSize, size_t gen) {
    dim3 grid, block;

    assert(pop != nullptr);

    //TODO set block and grid size more effectivelly
    block = dim3(BLOCK_SIZE);
    grid = dim3((popSize + BLOCK_SIZE - 1)/ BLOCK_SIZE);

    doTheMagicKernel<<<grid, block>>>(pop, popSize, indSize, gen);
    CCR(hipGetLastError());
}

CUDA_HOST   size_t*    getResults(int * pop, size_t popSize, size_t indSize, size_t * resSize) {
    dim3 grid, block;
    static size_t * d_res=  nullptr;
    size_t * h_res = nullptr;
    static size_t domSize = 0;

    // TODO set block & grid size
    block = dim3(BLOCK_SIZE);
    grid = dim3((domSize + BLOCK_SIZE - 1)/ BLOCK_SIZE);

    assert(pop != nullptr);

    if (domSize == 0 || cstrDomSize != domSize) {
        domSize = cstrDomSize;

        if (d_res) {
            CCR(hipFree((void*)d_res));
        }

        CCR(hipMalloc((void**)&d_res, sizeof(size_t) * domSize));
    }

    getResultsKernel<<<256, 32>>>(pop, popSize, indSize, cstrDomSize, d_res);
    CCR(hipGetLastError());
    CCR(hipFree((void*)pop));

    h_res = new size_t[cstrDomSize];
    *resSize = cstrDomSize;

    CCR(hipMemcpy(h_res, d_res, sizeof(size_t) * cstrDomSize, hipMemcpyDeviceToHost));

    return h_res;
}

/**
 * Randomly creates a population of candidates to evolve
 * @param popPtr the address where the population will be stored
 * @param popSize number of individuals
 * @param indSize size of an an individual
 */
CUDA_GLOBAL void    initPopulationKernel(int * popPtr, size_t popSize, size_t indSize) {
    size_t gtid = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(CURAND_SEED, gtid, 0, &cstrRandStates[gtid]);

    if (gtid < popSize){
        for (int i = 0; i<indSize; ++i){
            popPtr[indSize * gtid + i] = CurandInterval(hiprand(&cstrRandStates[gtid]), cstrDom[2 * i], cstrDom[(2 * i) + 1]);
            // Variable i is in [cstrDom[2i], cstrDom[2i + 1]]
        }
    }
}

/**
 * "Evolves" an individual (a set of values) to give it the lowest score
 * possible (We are looking for the worst possible candidat, to give the solver
 * some hint, where he must not search
 * @param pop the candidate population
 * @param popSize how many individual are in this population
 * @param indSize the individual's size (ints)
 * @param gen number of generations (epochs) before stopping
 */
CUDA_GLOBAL void    doTheMagicKernel(int * pop, size_t popSize, size_t indSize, size_t gen) {
    size_t gtid = blockIdx.x * blockDim.x + threadIdx.x;
    int old_fitness, cur_fitness;
    int * indiv = pop + (gtid * indSize); // points at the first element of our current individual
    int * child = new int[indSize];       // candidate for the next generation
    int mut_var = 0;                      // Mutated variable

    old_fitness = cstrValidate(indiv);

    for (int i = 0;  i < indSize; ++i){
        child[i] = indiv[i];
    }

    if (gtid < popSize){
        for (int epoch = 0; epoch < gen && old_fitness > 0; ++epoch){
            mut_var = CurandInterval(hiprand(&cstrRandStates[gtid]), 0, indSize - 1);
            child[mut_var] = CurandInterval(hiprand(&cstrRandStates[gtid]), cstrDom[2 * mut_var], cstrDom[(2 * mut_var) + 1]);
            cur_fitness = cstrValidate(child);
            if (cur_fitness < old_fitness){
                // We save the child
                indiv[mut_var] = child[mut_var];
                old_fitness = cur_fitness;
            }
            else{
                // We reset the child
                child[mut_var] = indiv[mut_var];
            }
        }
    }
}

/**
 * Counts how many occurences of a specific value we have, for a given variable
 * @param pop candidates population
 * @param popSize number of individuals in the population
 * @param indSize individual size (how many ints)
 * @param domSize the sum of each constraint's domain size
 * @param res number of occurencies of each variable's value
 */
CUDA_GLOBAL void    getResultsKernel(int * pop, size_t popSize, size_t indSize, size_t domSize, size_t* res) {
    size_t  gtid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t  sum = 0;
    size_t  idx = 0;          // variable's index
    int     val = cstrDom[0]; // value to test

    if (gtid < domSize) {
        // set val to the value we want to test
        for (size_t i = 0; (i < domSize) && (i < gtid); ++i) {
            ++val;

            if (val > cstrDom[2 * idx + 1]) {
                ++idx;
                val = cstrDom[2 * idx];
            }
        }

        for (size_t i = 0; i < popSize; ++i) {
            sum += (pop[i * indSize + idx] == val);
        }
        res[gtid] = sum;
    }
}

/**
 * Test each constraint on a candidate to evaluate it
 * @param c the candidate
 * @return how many constraints are satisfied
 */
CUDA_DEVICE int cstrValidate(int * c) {
    int satisfied = 0;
    for (size_t i = 0; cstrData[8 * i] != CSTR_NO && (8 * i) < CSTR_MAX_CSTR; ++i) {
        if (cstrTable[cstrData[8 * i]](cstrData + (8 * i) + 1, c)) {
            satisfied ++;
        }
    }

    return(satisfied);
}

CUDA_DEVICE bool cstrEq(uintptr_t * data, int * c) {
    size_t v0 = (size_t) data[0];
    int    val = uint2int((unsigned int) data[1]);

    return c[v0] == val;
}

CUDA_DEVICE bool cstrAndEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpAnd(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrAndNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpAnd(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrAndGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpAnd(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrAndGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpAnd(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrAndLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpAnd(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrAndLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpAnd(p0, c[v0], p1, c[v1]) <= c[v2];
}


CUDA_DEVICE bool cstrOrEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpOr(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrOrNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpOr(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrOrGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpOr(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrOrGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpOr(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrOrLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpOr(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrOrLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpOr(p0, c[v0], p1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrImpEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpImp(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrImpNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpImp(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrImpGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpImp(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrImpGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpImp(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrImpLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpImp(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrImpLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpImp(p0, c[v0], p1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrXorEQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpXor(p0, c[v0], p1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrXorNQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpXor(p0, c[v0], p1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrXorGQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpXor(p0, c[v0], p1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrXorGR(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpXor(p0, c[v0], p1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrXorLQ(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpXor(p0, c[v0], p1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrXorLE(uintptr_t * data, int * c) {
    bool p0 = (bool) data[0], p1 = (bool) data[2];
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpXor(p0, c[v0], p1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrPlusEQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpPlus(n0, c[v0], n1, c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrPlusNQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpPlus(n0, c[v0], n1, c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrPlusGQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpPlus(n0, c[v0], n1, c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrPlusGR(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpPlus(n0, c[v0], n1, c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrPlusLQ(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpPlus(n0, c[v0], n1, c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrPlusLE(uintptr_t * data, int * c) {
    int n0 = uint2int((unsigned int) data[0]), n1 = uint2int((unsigned int) data[2]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[3], v2 = (size_t) data[4];

    return OpPlus(n0, c[v0], n1, c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrTimesEQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return OpTimes(n, c[v0], c[v1]) == c[v2];
}

CUDA_DEVICE bool cstrTimesNQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return OpTimes(n, c[v0], c[v1]) != c[v2];
}

CUDA_DEVICE bool cstrTimesGQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return OpTimes(n, c[v0], c[v1]) > c[v2];
}

CUDA_DEVICE bool cstrTimesGR(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return OpTimes(n, c[v0], c[v1]) >= c[v2];
}

CUDA_DEVICE bool cstrTimesLQ(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return OpTimes(n, c[v0], c[v1]) < c[v2];
}

CUDA_DEVICE bool cstrTimesLE(uintptr_t * data, int * c) {
    int n = uint2int((unsigned int) data[0]);
    size_t v0 = (size_t) data[1], v1 = (size_t) data[2], v2 = (size_t) data[3];

    return OpTimes(n, c[v0], c[v1]) <= c[v2];
}

CUDA_DEVICE bool cstrLinearEQ(uintptr_t * data, int * c) {
    size_t vIdx = (size_t) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    size_t *v = cstrPoly + vIdx;
    int sum = 0;

    OpLinear(v, size, sum);
    return sum == c[v0];
}

CUDA_DEVICE bool cstrLinearNQ(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    OpLinear(v, size, sum);
    return sum != c[v0];
}

CUDA_DEVICE bool cstrLinearGQ(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    OpLinear(v, size, sum);
    return sum > c[v0];
}

CUDA_DEVICE bool cstrLinearGR(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    OpLinear(v, size, sum);
    return sum >= c[v0];
}

CUDA_DEVICE bool cstrLinearLQ(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    OpLinear(v, size, sum);
    return sum < c[v0];
}

CUDA_DEVICE bool cstrLinearLE(uintptr_t * data, int * c) {
    size_t * v = (size_t*) data[0], size = (size_t) data[1];
    size_t v0 = (size_t) data[2];
    int sum = 0;

    OpLinear(v, size, sum);
    return sum <= c[v0];
}
