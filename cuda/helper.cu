#include <cuda/helper.hh>

size_t * pushPolyToGPU(size_t * poly, size_t size) {
    size_t * ret;

    CCR(hipMalloc((void**)&ret, size * sizeof(size_t)));
    CCR(hipMemcpy(ret, poly, size * sizeof(size_t), hipMemcpyHostToDevice));

    return ret;
}
